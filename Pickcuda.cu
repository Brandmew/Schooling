
#include <hip/hip_runtime.h>
//#include "Globals.h"
//#include "Pickcuda.cuh"
//__global__ void AnimateCU(float timedelta, int cnt, int MiddleSpeed)
//{
//	int i = threadIdx.x;
//	if (i < cnt)
//	{
//		//Bird[i].Vel += ((MiddleSpeed - D3DXVec3Length(&Bird[i].Vel))*0.001F)*Bird[i].Vel;
//		Bird[i].Vel += Bird[i].Acc * timedelta;
//		Bird[i].Pos += Bird[i].Vel * timedelta;
//	}
//}
//
//cudaError CudaAnimate(Globs *pGlbs, float timeDelta)
//{
//	int NoBrds = pGlbs->BirdsCount;
//	int AveSpeed = (pGlbs->BirdTopVel + pGlbs->BirdBottomVel) / 2;
//	int NBlks = NoBrds / ThdsPerBlk;
//	AnimateCU <<< NBlks, ThdsPerBlk >>> (timeDelta, NoBrds, AveSpeed);
//	return cudaGetLastError();
//}
